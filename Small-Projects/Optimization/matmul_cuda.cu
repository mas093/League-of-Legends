#include <iostream>
#include <vector>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "kernel.cu"
#include "cu_array.h"
#include <math.h>

using namespace std;

int main()
{
	//Definitions
	int N = 500;
	int SIZE = N * N;

	//Allocate host memory
	vector<float> h_A;
	h_A.reserve(SIZE);
	vector<float> h_B;
	h_B.reserve(SIZE);
	vector<float> h_C;
	h_C.resize(SIZE);

	//Fill matrices
	for (int i = 0; i < N; i++) {
		for (int j = 0;j < N;j++)
		{
			h_A[i * N + j] = rand() % 100;
			h_B[i * N + j] = rand() % 100;
		}
	}

	//Allocate device memory
	cu_array<float> d_A(SIZE);
	cu_array<float> d_B(SIZE);
	cu_array<float> d_C(SIZE);

	d_A.set(&h_A[0], SIZE);
	d_B.set(&h_B[0], SIZE);



	//Measure elapsed time on the GPU (time.h is CPU)
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipDeviceSynchronize();
	hipEventRecord(start);
	
	matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
	
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	//d_C.get(&h_C[0], SIZE); Copy back does not work for large N


	cout << milliseconds << "ms" << endl;
	return 0;
}